#include "hip/hip_runtime.h"
#include "kernels.cuh"


void muli_vs(float *v, int s, int N) {
  int num_threads = min(N, MAX_THREADS_PER_BLOCK);
  int num_blocks = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
  k_muli_vs<<<num_blocks, num_threads>>>(v, s, N);
}

__global__ k_muli_vs(float *v, int s, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) return;

  v[idx] *= s;
}


void addi_vv(float *v1, const float *v2, float v1_coeff, float v2_coeff,
    int N) {
  if (v1_coeff != 1.0)
    muli_vs(v1, v1_coeff, N);

  hipblasSaxpy(handle, N, v2_coeff, v2, 1, v1, 1);
}


void subtensor1(float *dst, const float *src, const int *idxs, int N, int D,
    int idx_scal_shift, int idx_vec_shift_coeff, int *idx_vec_shift) {
  int num_threads = min(D, MAX_THREADS_PER_BLOCK);
  int num_blocks = min(N, MAX_BLOCKS);
  k_subtensor1<<<num_blocks, num_threads>>>(dst, src, idxs, N, D,
      idx_scal_shift, idx_vec_shift_coeff, idx_vec_shift);
}

__global__ void k_subtensor1(float *dst, const float *src, const int *idxs,
    int N, int D, int idx_scal_shift, int idx_vec_shift_coeff,
    int *idx_vec_shift) {
  for (int i0 = blockIdx.x; i0 < N; i0 += gridDim.x) {
    int src_idx = idxs[i0] + idx_scal_shift;
    src_idx += idx_vec_shift_coeff * idx_vec_shift[i0];

    int src_offset = src_idx * D;
    int dst_offset = i0 * D;
    for (int i1 = threadIdx.x; i1 < D; i1 += blockDim.x)
      dst[dst_offset + i1] = src[src_offset + i1];
  }
}


void set_subtensor1i_s(int *dst, int src, const int *idxs, int N,
    int idx_scal_shift, int idx_vec_shift_coeff, int *idx_vec_shift) {
  int num_threads = min(N, MAX_THREADS_PER_BLOCK);
  int num_blocks = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
  k_set_subtensor1i_s<<<num_blocks, num_threads>>>(
      dst, src, idxs, N, idx_scal_shift, idx_vec_shift_coeff, idx_vec_shift);
}

__global__ k_set_subtensor1i_s(int *dst, int src, const int *idxs, int N,
    int idx_scal_shift, int idx_vec_shift_coeff, int *idx_vec_shift) {
  int k_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) return;

  idx = idxs[k_idx] + idx_scal_shift;
  idx += idx_vec_shift_coeff * idx_vec_shift[k_idx];

  dst[idx] = src;
}
